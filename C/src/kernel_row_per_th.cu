#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>

#include "../headers/kernel_row_per_th.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line);


__global__ void gpu_kernel_1(int* col_indx,  int* csr_rows, int nnz, int rows, int* out_sum){
    // Indexing for block/threads
    int bid = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int tid = threadIdx.x + threadIdx.y*blockDim.x;
    int stride = blockDim.x*blockDim.y;

    if(bid >= rows){
        return;
    }
    // ------- Dynamic memory allocations -------
    extern __shared__ int sh_arr[];
    int* sh_sum = sh_arr;
    int* sh_row = &sh_arr[stride];
    int* sh_len = &sh_arr[stride*2];
    int* sh_cols = &sh_arr[stride*3];
    sh_sum[tid] = 0;
    // ------- Read main row -------
    /* The thread 0 of each wrap will read one
        element from the global memory and shuffle it
        to the rest.                                   
    */
    int start_row;
    int end_row;
    if(threadIdx.x == 0){
        start_row = csr_rows[bid]; 
        end_row = bid==(rows-1) ? nnz : csr_rows[bid+1];
    }
     
    start_row = __shfl_sync(0xffffffff, start_row, 0);
    end_row = __shfl_sync(0xffffffff, end_row, 0);
   
    int len = end_row - start_row;
    if(tid < len){
        sh_row[tid] = col_indx[start_row + tid];
    }
    

    __syncthreads();

    // ------- Read all the columns -------
    for(int i = 0; i < len; i++){
        int col = sh_row[i];
        start_row = csr_rows[col];
        end_row = col==(rows-1)? nnz : csr_rows[col+1];
        int templen = end_row - start_row;
        if(tid == 0)
            sh_len[i] = templen;
        if(tid < templen){
            sh_cols[i*stride + tid] = col_indx[start_row + tid];
        } 
    }

    __syncthreads();
    /* Each thread, will check every element of one column
        against the elements of the main row */
    int sum = 0;
    if(tid < len){
        int a = 0; 
        int b = 0;
        int blen = sh_len[tid];
        while(1){
            if(a == len || b == blen)
                break;
            int b1 = sh_row[a] == sh_cols[tid*stride + b];
            int b2 = sh_row[a] > sh_cols[tid*stride + b];
            int b3 = sh_row[a] < sh_cols[tid*stride + b];

            a = a + b1 + b3;
            b = b + b1 + b2;
            sum = sum + b1;
            
        }
    }
   __syncthreads();
    for(int w = 16; w > 0; w = w/2){
        sum += __shfl_down_sync(0xffffffff, sum, w, 32);
    }
    if(threadIdx.x == 0)
        sh_sum[threadIdx.y] = sum;
    __syncthreads();
    if(tid == 0){
        for(int i = 1; i < blockDim.y; i++){
            sh_sum[0] += sh_sum[i];
        } 
        out_sum[bid] = sh_sum[0];
    }
}

int CountTriangles_row_per_th(int rows, int nnz, int max_per_row, int* intg_csr_rows, int* col_indx){
    int *dev_cols, *dev_csr_rows;
    int* dev_sum;
    int* res_sum = (int*)malloc(rows*sizeof(int));
    gpuErrchk(hipMalloc((void**)&dev_cols, nnz*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_csr_rows, rows*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_sum, rows*sizeof(int)));

    gpuErrchk(hipMemcpy((void*)dev_cols, col_indx, nnz*sizeof(int),hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy((void*)dev_csr_rows, intg_csr_rows, rows*sizeof(int),hipMemcpyHostToDevice))
    int blocks = rows/(512*512) + 1;
    int thread_groups = 0;
    /* We need more threads than the maximum number of non zero elements per row */
    while(1){
        thread_groups += 1;
        if(thread_groups*32 >= max_per_row){
            break;
        }
    }
    int threads = 32*thread_groups;
    /* Each shared memory array must have a multiple of 32 elements to avoid
        bank conflicts */
    int shmem = (max_per_row + 3)*threads*sizeof(int);
    printf("smem threads %d %d %d\n",threads, shmem, sizeof(int));
    gpu_kernel_1<<<dim3(512,512,blocks),dim3(32, thread_groups,1),shmem>>>(dev_cols, dev_csr_rows, nnz, rows, dev_sum);
    gpuErrchk(hipMemcpy(res_sum, dev_sum, rows*sizeof(int),hipMemcpyDeviceToHost))
    int s = 0;
    for(int i = 0; i < rows; i++){
        s += res_sum[i];
    }

    return s;


}

inline void gpuAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}