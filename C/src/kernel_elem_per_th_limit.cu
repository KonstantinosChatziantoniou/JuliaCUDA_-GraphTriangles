#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "../headers/kernel_elem_per_th_limit.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line);


__global__ void gpu_kernel_3(int* col_indx,  int* csr_rows, int nnz, int rows,
                                int* out_sum, int max_per_row, int max_mem_nnz){
    int bid = blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y;
    int tid = threadIdx.x + threadIdx.y*blockDim.x;
    int stride = blockDim.x*blockDim.y;

    if(bid >= rows){
        return;
    }
    extern __shared__ int sh_arr[];
    int* sh_row = sh_arr;
    int* sh_sum = &sh_arr[max_mem_nnz];
    int* sh_cols = &sh_arr[max_mem_nnz + stride];
    sh_sum[tid] = 0;
    // Read main row -------------------------------------------
    int start_row = 1;
    int end_row = 2;
    if(threadIdx.x == 0){
        start_row = csr_rows[bid]; 
        end_row = bid==(rows-1) ? nnz : csr_rows[bid+1];
    }
     
    start_row = __shfl_sync(0xffffffff, start_row, 0);
    end_row = __shfl_sync(0xffffffff, end_row, 0);
   
    int len = end_row - start_row;
    for(int r = 0; r < len; r+=stride){
        if(tid+r < len){
            sh_row[tid+r] = col_indx[start_row + tid  + r];
        }
    }

    __syncthreads();
    /* Read the elements of the columns and stores them contigiously in
    memory. When there are enough elements (more than threads), each thread
    will take an element and compare it to all the elements of the main row */
    int total_elems = 0;
    int sum = 0;
    for(int i = 0; i < len; i++){
        int col = sh_row[i];
        start_row = csr_rows[col];
        end_row = col==(rows-1)? nnz : csr_rows[col+1];
        int temp_len = end_row-start_row;
        for(int r = 0; r < len; r+=stride){
            if(tid+r < temp_len){
                sh_cols[total_elems + tid + r] = col_indx[start_row + tid + r];
            }
        }
        total_elems += temp_len;
        __syncthreads();
        if(total_elems >= stride || i == (len-1)){
            for(int s = 0; s < total_elems; s += stride){
                for(int l = 0; l < len; l++){
                    if(tid + s < total_elems){
                        sum += (sh_row[(tid+l)%len] == sh_cols[s + tid]);
                    }
                }
            }
            total_elems = 0;
        }
    }
    

    __syncthreads();
    for(int w = 16; w > 0; w = w/2){
        sum += __shfl_down_sync(0xffffffff, sum, w, 32);
    }
    if(threadIdx.x == 0)
        sh_sum[threadIdx.y] = sum;
    __syncthreads();
    if(tid == 0){
        for(int i = 1; i < blockDim.y; i++){
            sh_sum[0] += sh_sum[i];
        } 
        out_sum[bid] = sh_sum[0];
    }
}

int CountTriangles_elem_per_th_limit(int rows, int nnz, int max_per_row, int* intg_csr_rows, int* col_indx, int thread_groups){
    int *dev_cols, *dev_csr_rows;
    int* dev_sum;
    int* res_sum = (int*)malloc(rows*sizeof(int));
    gpuErrchk(hipMalloc((void**)&dev_cols, nnz*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_csr_rows, rows*sizeof(int)));
    gpuErrchk(hipMalloc((void**)&dev_sum, rows*sizeof(int)));

    gpuErrchk(hipMemcpy((void*)dev_cols, col_indx, nnz*sizeof(int),hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy((void*)dev_csr_rows, intg_csr_rows, rows*sizeof(int),hipMemcpyHostToDevice))
    int blocks = rows/(512*512) + 1;
    //int thread_groups = 2;
    int threads = 32*thread_groups;
    int main_rows_smem = 0;
    if(max_per_row%32 == 0){
        main_rows_smem = max_per_row;
    }else{
        main_rows_smem = 32*(max_per_row/32 + 1);
    } 
    int cols_smem = threads > max_per_row ? threads : max_per_row;
    int shmem = (main_rows_smem + threads + 2*cols_smem)*sizeof(int);
    printf("SHMEME %d maxpr %d\n", shmem, main_rows_smem);

    gpu_kernel_3<<<dim3(512,512,blocks),dim3(32,thread_groups,1),shmem>>>(dev_cols, dev_csr_rows, nnz, rows,
                 dev_sum, max_per_row ,main_rows_smem);
    gpuErrchk(hipMemcpy(res_sum, dev_sum, rows*sizeof(int),hipMemcpyDeviceToHost))
    int s = 0;
    for(int i = 0; i < rows; i++){
        s += res_sum[i];
    }

    return s;


}

inline void gpuAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}